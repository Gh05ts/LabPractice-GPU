#include "hip/hip_runtime.h"
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include "support.h"

__constant__ float M_c[FILTER_SIZE][FILTER_SIZE];

__global__ void convolution(hipTextureObject_t N, Matrix P)
{
    /********************************************************************
    Determine input and output indexes of each thread
    Load a tile of the input image to shared memory
    Apply the filter on the input image tile
    Write the compute values to the output image at the correct indexes
    ********************************************************************/

    constexpr int filter_rad = (FILTER_SIZE - 1) / 2;

    // INSERT KERNEL CODE HERE
    int outCol = blockIdx.x * blockDim.x + threadIdx.x;
    int outRow = blockIdx.y * blockDim.y + threadIdx.y;

    if (outCol >= P.width || outRow >= P.height) return;

    float Pvalue = 0.0f;
    for (int fy = -filter_rad; fy <= filter_rad; fy++) {
        for (int fx = -filter_rad; fx <= filter_rad; fx++) {
            float pixel = tex2D<float>(N, outCol + fx, outRow + fy);
            // float weight = M_c[(fy + filter_rad) * FILTER_SIZE + (fx + filter_rad)];
            float weight = M_c[fy + filter_rad][fx + filter_rad];
            Pvalue += pixel * weight;
        }
    }    
    P.elements[outRow * P.width + outCol] = Pvalue;
}
