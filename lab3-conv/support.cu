/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdlib.h>
#include <stdio.h>

#include "support.h"

hipArray* allocateDeviceArray(unsigned height, unsigned width) {
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipArray* cuArray;
    hipMallocArray(&cuArray, &channelDesc, width, height, hipArrayDefault);
    return cuArray;
}

void printMatrix(Matrix M) {
    for(int i = 0; i < M.height; i++) {
        int x = i * M.width;
        for(int j = 0; j < M.width; j++) {
            printf("%f, ", M.elements[x + j]);
        }
        printf("\n");
    }
}

hipTextureObject_t allocateTex(hipArray *cuArray, Matrix h_input, unsigned height, unsigned width) {
    hipMemcpy2DToArray(
        cuArray,
        0, 0,
        h_input.elements,
        width * sizeof(float),
        width * sizeof(float),
        height,
        hipMemcpyHostToDevice
    );

    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    hipTextureDesc texDesc = {};
    texDesc.addressMode[0] = hipAddressModeBorder;
    texDesc.addressMode[1] = hipAddressModeBorder;
    texDesc.borderColor[0] = 0.0f;
    texDesc.borderColor[1] = 0.0f;
    texDesc.borderColor[2] = 0.0f;
    texDesc.borderColor[3] = 0.0f;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    hipTextureObject_t texObj = 0;
    hipCreateTextureObject(&texObj, &resDesc, &texDesc, nullptr);

    return texObj;
}

Matrix allocateMatrix(unsigned height, unsigned width)
{
    Matrix mat;
    mat.height = height;
    mat.width = mat.pitch = width;
    hipHostAlloc(&mat.elements, height * width * sizeof(float), hipHostMallocDefault);
    if (mat.elements == NULL)
        FATAL("Unable to allocate host");

    return mat;
}

void initMatrix(Matrix mat)
{
    for (unsigned int x = 0; x < mat.height * mat.width; x++) {
        mat.elements[x] = (rand() % 100) / 100.00;
    }
}

Matrix allocateDeviceMatrix(unsigned height, unsigned width)
{
    Matrix mat;
    hipError_t cuda_ret;

    mat.height = height;
    mat.width = mat.pitch = width;
    cuda_ret = hipMalloc((void **)&(mat.elements), height * width * sizeof(float));
    if (cuda_ret != hipSuccess)
        FATAL("Unable to allocate device memory");

    return mat;
}

Matrix allocateDeviceMatrixPitched(unsigned height, unsigned width) {
    Matrix mat;
    hipError_t cuda_ret;

    mat.height = height;
    mat.width = width;
    size_t pitch;
    cuda_ret = hipMallocPitch(&mat.elements, &pitch, width*sizeof(float), height);
    if (cuda_ret != hipSuccess)
        FATAL("Unable to allocate device memory");

    mat.pitch = pitch;
    return mat;
}

void copyToDeviceMatrix(Matrix dst, Matrix src)
{
    hipError_t cuda_ret;
    cuda_ret = hipMemcpy(dst.elements, src.elements, src.height * src.width * sizeof(float), hipMemcpyHostToDevice);
    if (cuda_ret != hipSuccess)
        FATAL("Unable to copy to device");
}

void copyFromDeviceMatrix(Matrix dst, Matrix src)
{
    hipError_t cuda_ret;
    // hipMemcpy2D(dst.elements, , src.elements)
    cuda_ret = hipMemcpy(dst.elements, src.elements, src.height * src.width * sizeof(float), hipMemcpyDeviceToHost);
    if (cuda_ret != hipSuccess)
        FATAL("Unable to copy from device");
}

void verify(Matrix M, Matrix N, Matrix P)
{

    const float relativeTolerance = 1e-6;

    for (int row = 0; row < N.height; ++row)
    {
        for (int col = 0; col < N.width; ++col)
        {
            float sum = 0.0f;
            for (int i = 0; i < M.height; ++i)
            {
                for (int j = 0; j < M.width; ++j)
                {
                    int iN = row - M.height / 2 + i;
                    int jN = col - M.width / 2 + j;
                    if (iN >= 0 && iN < N.height && jN >= 0 && jN < N.width)
                    {
                        sum += M.elements[i * M.width + j] * N.elements[iN * N.width + jN];
                    }
                }
            }
            float relativeError = (sum - P.elements[row * P.width + col]) / sum;
            if (relativeError > relativeTolerance || relativeError < -relativeTolerance)
            {
                printf("TEST FAILED\n\n");
                exit(0);
            }
        }
    }
    printf("TEST PASSED\n\n");
}

void freeMatrix(Matrix mat)
{
    hipFree(mat.elements);
    // free(mat.elements);
    mat.elements = NULL;
}

void freeDeviceMatrix(Matrix mat)
{
    hipFree(mat.elements);
    mat.elements = NULL;
}

void startTime(Timer *timer)
{
    gettimeofday(&(timer->startTime), NULL);
}

void stopTime(Timer *timer)
{
    gettimeofday(&(timer->endTime), NULL);
}

float elapsedTime(Timer timer)
{
    return ((float)((timer.endTime.tv_sec - timer.startTime.tv_sec) + (timer.endTime.tv_usec - timer.startTime.tv_usec) / 1.0e6));
}
