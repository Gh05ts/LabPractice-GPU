
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

__global__ 
void vecAddKernel(float *A, float *B, float *C, int n)
{
	// Calculate global thread index based on the block and thread indices ----
	//INSERT KERNEL CODE HERE
	int i = threadIdx.x + blockDim.x * blockIdx.x;

	// Use global index to determine which elements to read, add, and write ---
	//INSERT KERNEL CODE HERE, BE CAREFUL FOR CORNER CASE!!!
	if (i < n) {
		C[i] = A[i] + B[i];
	}
}

__global__ 
void image2grayKernel(const float *in, float *out, int height, int width)
{
	// Calculate global thread index based on the block and thread indices ----
	//INSERT KERNEL CODE HERE
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	if(col < width && row < height) {
		int grayOffset = row*width + col;

		int rgbOffset = grayOffset*3; // channels = 3
		unsigned char r = in[rgbOffset];
		unsigned char g = in[rgbOffset + 1];
		unsigned char b = in[rgbOffset + 2];

		out[grayOffset] = 0.144f*r + 0.587f*g + 0.299f*b;
	}
	// Use global index to determine which elements to read, add, and write ---
	//INSERT KERNEL CODE HERE, BE CAREFUL FOR CORNER CASE!!!
	
}

__global__
__launch_bounds__(256)
void vecAdd4(const float4* __restrict__ A, const float4* __restrict__ B, float4* __restrict__ C, const int vectorizedN) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < vectorizedN) {
		float4 a = A[idx];
		float4 b = B[idx];
		C[idx] = make_float4(a.x + b.x, a.y + b.y, a.z + b.z, + a.w + b.w);
	}

}

__global__
__launch_bounds__(256)
void image2grayKernelOpt(const float3* __restrict__ in, float* __restrict__ out, const int height, const int width)
{
	// Calculate global thread index based on the block and thread indices ----
	//INSERT KERNEL CODE HERE
	int col = blockIdx.x*blockDim.x + threadIdx.x; // * 4
	// int row = blockIdx.y*blockDim.y 

	if(col < width * height) {
		// int grayOffset = row*width + col;
		// int colOffset = 
		// int rgbOffset = grayOffset*3; // channels = 3
		// unsigned char r = in[rgbOffset];
		// unsigned char g = in[rgbOffset + 1];
		// unsigned char b = in[rgbOffset + 2];

		// printf("%f\n", in[col].x);
		// printf("%f\n", in[col].y);
		// printf("%f\n", in[col].z);
		// out[grayOffset] = 0.144f*in[grayOffset].x+ 0.587f*in[grayOffset].y + 0.299f*in[grayOffset].z;
		// make_float4();
		out[col] = 0.144f*in[col].x+ 0.587f*in[col].y + 0.299f*in[col].z;
		// out[col + 1] = 0.144f*in[col + 1].x+ 0.587f*in[col + 1].y + 0.299f*in[col + 1].z;
		// out[col + 1] = 0.144f*in[col + 2].x+ 0.587f*in[col + 2].y + 0.299f*in[col + 2].z;
		// out[col + 1] = 0.144f*in[col + 3].x+ 0.587f*in[col + 3].y + 0.299f*in[col + 3].z;
		// printf("%f/n/n/n", out[col]);
	}
	// Use global index to determine which elements to read, add, and write ---
	//INSERT KERNEL CODE HERE, BE CAREFUL FOR CORNER CASE!!!
	
}
