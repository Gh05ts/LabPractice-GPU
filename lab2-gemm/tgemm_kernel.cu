/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

// Feel free to use other numbers for best performance
#define TILE_SIZE 32
#define BLOCK_SIZE 32

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float *C)
{

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     * Use shared memory for tiling
     *
     ********************************************************************/

    // INSERT KERNEL CODE HERE
    
    __shared__ float sA[TILE_SIZE][TILE_SIZE];
    __shared__ float sB[TILE_SIZE][TILE_SIZE];

    int local_x = threadIdx.x;
    int local_y = threadIdx.y;

    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    int row = blockRow * TILE_SIZE + local_y;
    int col = blockCol * TILE_SIZE + local_x;

    float sum = 0.0;
    int tiles = k / TILE_SIZE;

    for(int t = 0; t < tiles; t++) {        
        int a_col = t * TILE_SIZE + local_x;
        int a_index = row * k + a_col;
        sA[local_y][local_x] = A[a_index];

        int b_row = t * TILE_SIZE + local_y;
        int b_col = blockCol * TILE_SIZE + local_x;
        int b_index = b_row * n + b_col;
        sB[local_y][local_x] = B[b_index];
        __syncthreads();

        #pragma unroll
        for(int i = 0; i < TILE_SIZE; i++) {
            sum += sA[local_y][i] * sB[i][local_x];
        }

        __syncthreads();
    }
    C[row*n+col] = sum;
}

void basicSgemm(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc, int testRound)
{
    if ((transa != 'N') && (transa != 'n'))
    {
        printf("unsupported value of 'transa'\n");
        return;
    }

    if ((transb != 'N') && (transb != 'n'))
    {
        printf("unsupported value of 'transb'\n");
        return;
    }

    if ((alpha - 1.0f > 1e-10) || (alpha - 1.0f < -1e-10))
    {
        printf("unsupported value of alpha\n");
        return;
    }

    if ((beta - 0.0f > 1e-10) || (beta - 0.0f < -1e-10))
    {
        printf("unsupported value of beta\n");
        return;
    }

    // Initialize thread block and kernel grid dimensions ----------------------
    // INSERT CODE HERE
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((n + dimBlock.x - 1) / dimBlock.x, (m + dimBlock.y - 1)/ dimBlock.y);    

    for (int i = 0; i < testRound; i++) {
        // Invoke CUDA kernel --------------------------------------------------
        // INSERT CODE HERE
        mysgemm<<<dimGrid, dimBlock>>>(m, n, k, A, B, C);
        hipDeviceSynchronize();
    }
}
